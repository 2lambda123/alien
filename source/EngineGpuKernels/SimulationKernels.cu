#include "hip/hip_runtime.h"
﻿#include "SimulationKernels.cuh"
#include "FlowFieldKernels.cuh"
#include "ClusterProcessor.cuh"
#include "CellFunctionProcessor.cuh"
#include "NerveProcessor.cuh"
#include "NeuronProcessor.cuh"
#include "ConstructorProcessor.cuh"
#include "AttackerProcessor.cuh"
#include "TransmitterProcessor.cuh"
#include "MuscleProcessor.cuh"

__global__ void cudaNextTimestep_prepare(SimulationData data, SimulationResult result)
{
    data.prepareForNextTimestep();
}

__global__ void cudaNextTimestep_physics_substep1(SimulationData data)
{
    CellProcessor::init(data);
    CellProcessor::updateMap(data);
    CellProcessor::radiation(data);  //do not use ParticleProcessor in this kernel
    CellProcessor::clearDensityMap(data);
}

__global__ void cudaNextTimestep_physics_substep2(SimulationData data)
{
    CellProcessor::collisions(data);
    CellProcessor::fillDensityMap(data);

    ParticleProcessor::updateMap(data);
}

__global__ void cudaNextTimestep_physics_substep3(SimulationData data)
{
    CellProcessor::checkForces(data);
    CellProcessor::updateVelocities(data);

    ParticleProcessor::movement(data);
    ParticleProcessor::collision(data);
}

__global__ void cudaNextTimestep_physics_substep4(SimulationData data)
{
    CellProcessor::verletPositionUpdate(data);
    CellProcessor::checkConnections(data);
}

__global__ void cudaNextTimestep_physics_substep5(SimulationData data, bool considerAngles)
{
    CellProcessor::calcConnectionForces(data, considerAngles);
}

__global__ void cudaNextTimestep_physics_substep6(SimulationData data)
{
    CellProcessor::verletVelocityUpdate(data);
}

__global__ void cudaNextTimestep_cellFunction_prepare_substep1(SimulationData data)
{
    CellFunctionProcessor::aging(data);
}

__global__ void cudaNextTimestep_cellFunction_prepare_substep2(SimulationData data)
{
    CellProcessor::constructionStateTransition(data);
    CellFunctionProcessor::collectCellFunctionOperations(data);
}

__global__ void cudaNextTimestep_cellFunction_nerve(SimulationData data, SimulationResult result)
{
    NerveProcessor::process(data, result);
}

__global__ void cudaNextTimestep_cellFunction_neuron(SimulationData data, SimulationResult result)
{
    NeuronProcessor::process(data, result);
}

__global__ void cudaNextTimestep_cellFunction_constructor(SimulationData data, SimulationResult result)
{
    ConstructorProcessor::process(data, result);
}

__global__ void cudaNextTimestep_cellFunction_attacker(SimulationData data, SimulationResult result)
{
    AttackerProcessor::process(data, result);
}

__global__ void cudaNextTimestep_cellFunction_transmitter(SimulationData data, SimulationResult result)
{
    TransmitterProcessor::process(data, result);
}

__global__ void cudaNextTimestep_cellFunction_muscle(SimulationData data, SimulationResult result)
{
    MuscleProcessor::process(data, result);
}

__global__ void cudaNextTimestep_physics_substep7_innerFriction(SimulationData data)
{
    CellProcessor::applyInnerFriction(data);
}

__global__ void cudaNextTimestep_physics_substep8(SimulationData data)
{
    CellFunctionProcessor::resetFetchedActivities(data);
    CellProcessor::applyFriction(data);
    CellProcessor::decay(data);
}

__global__ void cudaNextTimestep_structuralOperations_substep1(SimulationData data)
{
    data.structuralOperations.saveNumEntries();
}

__global__ void cudaNextTimestep_structuralOperations_substep2(SimulationData data)
{
    CellConnectionProcessor::processConnectionsOperations(data);
}

__global__ void cudaNextTimestep_structuralOperations_substep3(SimulationData data)
{
    ParticleProcessor::transformation(data);

    CellConnectionProcessor::processDelCellOperations(data);
}

__global__ void cudaInitClusterData(SimulationData data)
{
    ClusterProcessor::initClusterData(data);
}

__global__ void cudaFindClusterIteration(SimulationData data)
{
    ClusterProcessor::findClusterIteration(data);
}

__global__ void cudaFindClusterBoundaries(SimulationData data)
{
    ClusterProcessor::findClusterBoundaries(data);
}

__global__ void cudaAccumulateClusterPosAndVel(SimulationData data)
{
    ClusterProcessor::accumulateClusterPosAndVel(data);
}

__global__ void cudaAccumulateClusterAngularProp(SimulationData data)
{
    ClusterProcessor::accumulateClusterAngularProp(data);
}

__global__ void cudaApplyClusterData(SimulationData data)
{
    ClusterProcessor::applyClusterData(data);
}


//This is the only kernel that uses dynamic parallelism.
//When it is removed, performance drops by about 20% for unknown reasons.
__global__ void nestedDummy() {}
__global__ void dummy()
{
    nestedDummy<<<1, 1>>>();
}
