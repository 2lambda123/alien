﻿#include "SimulationKernelsLauncher.cuh"

#include "SimulationKernels.cuh"
#include "FlowFieldKernels.cuh"
#include "GarbageCollectorKernelsLauncher.cuh"
#include "DebugKernels.cuh"

_SimulationKernelsLauncher::_SimulationKernelsLauncher()
{
    _garbageCollector = std::make_shared<_GarbageCollectorKernelsLauncher>();
}

void _SimulationKernelsLauncher::calcTimestep(Settings const& settings, SimulationData const& data, SimulationResult const& result)
{
    auto const gpuSettings = settings.gpuSettings;
    KERNEL_CALL_1_1(cudaNextTimestep_prepare, data, result);
    if (settings.simulationParameters.numSpots > 0) {
        KERNEL_CALL(cudaApplyFlowFieldSettings, data);
    }

    //not all kernels need to be executed in each time step for performance reasons
    bool considerForcesFromAngleDifferences = (data.timestep % 3 == 0);
    bool considerInnerFriction = (data.timestep % 3 == 0);
    bool considerRigidityUpdate = (data.timestep % 3 == 0);

    KERNEL_CALL(cudaNextTimestep_physics_init, data);
    KERNEL_CALL(cudaNextTimestep_physics_substep1, data);
    KERNEL_CALL(cudaNextTimestep_physics_substep2, data);
    KERNEL_CALL(cudaNextTimestep_physics_substep3, data);
    KERNEL_CALL(cudaNextTimestep_physics_substep5, data, considerForcesFromAngleDifferences);
    KERNEL_CALL(cudaNextTimestep_physics_substep4, data);
    KERNEL_CALL(cudaNextTimestep_physics_substep5, data, considerForcesFromAngleDifferences);
    KERNEL_CALL(cudaNextTimestep_physics_substep6, data);

    //cell functions
    KERNEL_CALL(cudaNextTimestep_cellFunction_prepare_substep1, data);
    KERNEL_CALL(cudaNextTimestep_cellFunction_prepare_substep2, data);
    KERNEL_CALL(cudaNextTimestep_cellFunction_nerve, data, result);
    KERNEL_CALL(cudaNextTimestep_cellFunction_neuron, data, result);
    KERNEL_CALL(cudaNextTimestep_cellFunction_constructor, data, result);
    KERNEL_CALL(cudaNextTimestep_cellFunction_attacker, data, result);
    KERNEL_CALL(cudaNextTimestep_cellFunction_transmitter, data, result);
    KERNEL_CALL(cudaNextTimestep_cellFunction_muscle, data, result);
    KERNEL_CALL(cudaNextTimestep_cellFunction_sensor, data, result);

    if (considerInnerFriction) {
        KERNEL_CALL(cudaNextTimestep_physics_substep7_innerFriction, data);
    }
    KERNEL_CALL(cudaNextTimestep_physics_substep8, data);

    if (considerRigidityUpdate && isRigidityUpdateEnabled(settings)) {
        KERNEL_CALL(cudaInitClusterData, data);
        KERNEL_CALL(cudaFindClusterIteration, data);  //3 iterations should provide a good approximation
        KERNEL_CALL(cudaFindClusterIteration, data);
        KERNEL_CALL(cudaFindClusterIteration, data);
        KERNEL_CALL(cudaFindClusterBoundaries, data);
        KERNEL_CALL(cudaAccumulateClusterPosAndVel, data);
        KERNEL_CALL(cudaAccumulateClusterAngularProp, data);
        KERNEL_CALL(cudaApplyClusterData, data);
    }
    KERNEL_CALL_1_1(cudaNextTimestep_structuralOperations_substep1, data);
    KERNEL_CALL(cudaNextTimestep_structuralOperations_substep2, data);
    KERNEL_CALL(cudaNextTimestep_structuralOperations_substep3, data);

    _garbageCollector->cleanupAfterTimestep(settings.gpuSettings, data);
}

bool _SimulationKernelsLauncher::isRigidityUpdateEnabled(Settings const& settings) const
{
    for (int i = 0; i < settings.simulationParameters.numSpots; ++i) {
        if (settings.simulationParameters.spots[i].values.rigidity != 0) {
            return true;
        }
    }
    return settings.simulationParameters.baseValues.rigidity != 0;
}
