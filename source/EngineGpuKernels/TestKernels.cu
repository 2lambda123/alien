#include "hip/hip_runtime.h"
#include "TestKernels.cuh"

#include "MutationProcessor.cuh"

__global__ void cudaMutateNeuronData(SimulationData data, uint64_t cellId)
{
    auto& cells = data.objects.cellPointers;
    auto partition = calcAllThreadsPartition(cells.getNumEntries());

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        auto& cell = cells.at(index);
        if (cell->id == cellId) {
            MutationProcessor::mutateNeuronData(data, cell);
        }
    }
}

__global__ void cudaMutateData(SimulationData data, uint64_t cellId)
{
    auto& cells = data.objects.cellPointers;
    auto partition = calcAllThreadsPartition(cells.getNumEntries());

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        auto& cell = cells.at(index);
        if (cell->id == cellId) {
            MutationProcessor::mutateData(data, cell);
        }
    }
}
